#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cfloat>

// Error checking
#define GPU_CHECK(call) do { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(status) << std::endl; \
        exit(1); \
    } \
} while(0)

// Timer implementation for benchmarking
#ifdef __MACH__
#include <mach/mach.h>
#include <mach/mach_time.h>
static double getTime() {
    static mach_timebase_info_data_t info;
    static double conversion = 0.0;
    if (conversion == 0.0) {
        mach_timebase_info(&info);
        conversion = (info.numer / info.denom) / 1e9;
    }
    return conversion * mach_absolute_time();
}
#else
#include <ctime>
static double getTime() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}
#endif

__global__ void assignClusters(
    const float* points, const float* medoids, int* assignments, int* changed, const int N, const int K, const int D) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    float minDist = FLT_MAX;
    int bestCluster = -1;

    for (int k = 0; k < K; k++) {
        float dist = 0.0f;
        for (int d = 0; d < D; d++) {
            float diff = points[idx * D + d] - medoids[k * D + d];
            dist += diff * diff;
        }
        if (dist < minDist) {
            minDist = dist;
            bestCluster = k;
        }
    }

    if (assignments[idx] != bestCluster) {
        assignments[idx] = bestCluster;
        *changed = 1;
    }
}

__global__ void updateMedoids(
    const float* points, const int* assignments, float* medoids, const int N, const int K, const int D) {
    const int k = blockIdx.x;
    if (k >= K) return;

    __shared__ float bestDist;
    __shared__ int bestPoint;

    if (threadIdx.x == 0) {
        bestDist = FLT_MAX;
        bestPoint = -1;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        if (assignments[i] != k) continue;

        float totalDist = 0.0f;
        for (int j = 0; j < N; j++) {
            if (assignments[j] != k) continue;

            float dist = 0.0f;
            for (int d = 0; d < D; d++) {
                float diff = points[i * D + d] - points[j * D + d];
                dist += diff * diff;
            }
            totalDist += sqrtf(dist);
        }

        atomicMin((int*)&bestDist, __float_as_int(totalDist));
        if (__float_as_int(totalDist) == (int)bestDist) {
            bestPoint = i;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0 && bestPoint >= 0) {
        for (int d = 0; d < D; d++) {
            medoids[k * D + d] = points[bestPoint * D + d];
        }
    }
}

class FastKMedoids {
    int N, D, K;
    std::vector<float> points;
    std::vector<float> medoids;
    std::vector<int> assignments;

public:
    FastKMedoids(const char* filename, int k) : K(k) {
        std::ifstream fin(filename);
        fin >> N >> D;
        points.resize(N * D);
        for (float& x : points) fin >> x;
        medoids.assign(points.begin(), points.begin() + K * D);
        assignments.assign(N, -1);
    }

    void run(int threadsPerBlock) {
        float *d_points, *d_medoids;
        int *d_assignments, *d_changed;

        GPU_CHECK(hipMalloc(&d_points, N * D * sizeof(float)));
        GPU_CHECK(hipMalloc(&d_medoids, K * D * sizeof(float)));
        GPU_CHECK(hipMalloc(&d_assignments, N * sizeof(int)));
        GPU_CHECK(hipMalloc(&d_changed, sizeof(int)));

        GPU_CHECK(hipMemcpy(d_points, points.data(), N * D * sizeof(float), hipMemcpyHostToDevice));
        GPU_CHECK(hipMemcpy(d_medoids, medoids.data(), K * D * sizeof(float), hipMemcpyHostToDevice));

        int pointBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
        const int maxIter = 20;
        double startTime = getTime();

        for (int iter = 0; iter < maxIter; iter++) {
            int changed = 0;
            GPU_CHECK(hipMemcpy(d_changed, &changed, sizeof(int), hipMemcpyHostToDevice));
            assignClusters<<<pointBlocks, threadsPerBlock>>>(d_points, d_medoids, d_assignments, d_changed, N, K, D);
            GPU_CHECK(hipMemcpy(&changed, d_changed, sizeof(int), hipMemcpyDeviceToHost));
            if (!changed && iter > 0) break;
            updateMedoids<<<K, threadsPerBlock>>>(d_points, d_assignments, d_medoids, N, K, D);
        }

        double endTime = getTime();
        printf("k-medoids clustering time: %.4fs\n", endTime - startTime);

        GPU_CHECK(hipMemcpy(assignments.data(), d_assignments, N * sizeof(int), hipMemcpyDeviceToHost));
        GPU_CHECK(hipMemcpy(medoids.data(), d_medoids, K * D * sizeof(float), hipMemcpyDeviceToHost));

        // 메도이드 출력
        std::ofstream medoids_file("medoids.txt");
        for (int k = 0; k < K; k++) {
            for (int d = 0; d < D; d++) {
                medoids_file << medoids[k * D + d] << " ";
            }
            medoids_file << std::endl;
        }
        medoids_file.close();

        // 클러스터 할당 출력
        std::ofstream clusters_file("clusters.txt");
        for (int i = 0; i < N; i++) {
            clusters_file << assignments[i] << std::endl;
        }
        clusters_file.close();

        hipFree(d_points);
        hipFree(d_medoids);
        hipFree(d_assignments);
        hipFree(d_changed);
    }
};

int main(int argc, char** argv) {
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <K> <num_blocks> <threads_per_block>\n";
        return 1;
    }

    try {
        FastKMedoids clustering(argv[1], std::stoi(argv[2]));
        clustering.run(std::stoi(argv[4]));
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << '\n';
        return 1;
    }
    return 0;
}
